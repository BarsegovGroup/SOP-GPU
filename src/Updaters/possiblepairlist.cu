/*
 * possiblepairlist.cu
 *
 *  Created on: Apr 8, 2010
 *      Author: zhmurov
 */
#include "../gsop.cuh"
#include "possiblepairlist.h"

#include "possiblepairlist_kernel.cu"

void createPossiblepairlistUpdater(){
	updaters[updatersCount] = new PossiblepairList();
	updatersCount++;
}

/*
 * Building map of possible long-range contacts
 * (i.e. all pairs excluding native and covalent)
 * (for map description see initCovalent())
 */
PossiblepairList::PossiblepairList(){
	this->name = "Possiblepairlist";
	this->frequency = getIntegerParameter("possiblepairs_freq", 100000, 1);
	printf("Initializing possible pairs list generator...\n");
	this->blockSize = getIntegerParameter(POSSIBLEPAIRS_BLOCK_SIZE_STRING, gsop.blockSize, 1);
	this->blockNum = gsop.aminoCount/this->blockSize + 1;
	this->pairsThreshold = getFloatParameter(POSSIBLEPAIRS_CUTOFF_STRING, DEFAULT_POSSIBLEPAIRS_CUTOFF, 1);

    this->updateParametersOnGPU();

	if(deviceProp.major == 2){ // TODO: >= 2
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(generate_possiblepairs), hipFuncCachePreferL1);
	}
}

PossiblepairList::~PossiblepairList(){

}

void PossiblepairList::update(){
	if(step % this->frequency == 0){
		//printf("Regenerating the list of possible pairs...");
		generate_possiblepairs<<<this->blockNum, this->blockSize>>>();
		checkCUDAError();
		//printf("done.\n");
	}
}

void PossiblepairList::updateParametersOnGPU(){
    hc_possiblepairList.pairsThreshold = this->pairsThreshold;
	hipMemcpyToSymbol(HIP_SYMBOL(c_possiblepairList), &hc_possiblepairList, sizeof(PossiblepairListConstant), 0, hipMemcpyHostToDevice);
	checkCUDAError();
}


#include "hip/hip_runtime.h"
#include "../gsop.cuh"
/*
 * indentation_kernel.cu
 *
 *  Created on: Apr 9, 2010
 *      Author: zhmurov
 */

struct IndentationConstant {
	float3 tipCoord;
	float3 chipCoord;
	float3 chipCoord0;
	float3 direction;
	float V;
	float dx;
	int moveSurface;
	int fixTransversal;
	float tipRadius;
	float cantileverKs;
	float tipa6;
	float tipel;
	float surfa6;
	float surfel;
	float tipAprime;
	float tipBprime;
	float surfAprime;
	float surfBprime;
	float tipZeta;
	float3 micaN;
	float3 micaR0;
	float3 micaR;
	float4* h_tipForces;
	float4* d_tipForces;
	float4 tipForce;
	FILE* out;
	long int retractionStep;
	int showTipSurface;
	int surfaceSize;
	int surfaceBeadsCount;
	float surfaceStep;
	float3* surfacePointsR0;
	float4* h_surfacePointsCoord;
	float4* d_surfacePointsCoord;

	float pairsCutoff2;
	int* h_micaListCounts;
	int* d_micaListCounts;
	int* h_micaList;
	int* d_micaList;

	int outputFreq;
	float3 cantileverVector;
	float4 fav;
	float3 tipCoordAv;
	float3 chipCoordAv;
	float kDeltaXAv;
};

IndentationConstant hc_indentation;
__device__ __constant__ IndentationConstant c_indentation;

__global__ void indentation_kernel(){
	int d_i = blockIdx.x*blockDim.x + threadIdx.x;
	if(d_i < c_gsop.aminoCount){
		float4 coord = c_gsop.d_coord[d_i];
		float4 f = c_gsop.d_forces[d_i];
		float4 f_in = c_indentation.d_tipForces[d_i];//make_float4(0.0, 0.0, 0.0, 0.0);
		float4 df;
		float4 dr;
		dr.x =  coord.x - c_indentation.tipCoord.x;
		dr.y =  coord.y - c_indentation.tipCoord.y;
		dr.z =  coord.z - c_indentation.tipCoord.z;
		dr.w = sqrtf(dr.x*dr.x + dr.y*dr.y + dr.z*dr.z);
		float r2 = dr.w - c_indentation.tipRadius;
		r2 = 1.0f/r2;
		r2 = r2*r2;
		float r6 = r2*r2*r2;
		df.w = c_indentation.tipAprime*r6 + c_indentation.tipBprime;
		df.w = df.w*r6/(dr.w*(dr.w - c_indentation.tipRadius));
		df.x = dr.x*df.w;
		df.y = dr.y*df.w;
		df.z = dr.z*df.w;
		/*dr.w = sqrt(dr.x*dr.x+dr.y*dr.y+dr.z*dr.z);
		df.w = dr.w - c_indentation.tipRadius;
		df.w = powf(df.w, 7);
		//if(dr.w < c_pairs.pairsCutoff2){
			df.w = c_indentation.a6/df.w;
			df.w = -6.0*c_indentation.el*df.w/dr.w;
			df.x = dr.x*df.w;
			df.y = dr.y*df.w;
			df.z = dr.z*df.w;
		//}*/
		f_in.x += df.x;
		f_in.y += df.y;
		f_in.z += df.z;
		c_indentation.d_tipForces[d_i] = f_in;

		dr.x = coord.x - c_indentation.micaR.x;
		dr.y = coord.y - c_indentation.micaR.y;
		dr.z = coord.z - c_indentation.micaR.z;
		dr.w = dr.x*c_indentation.micaN.x + dr.y*c_indentation.micaN.y + dr.z*c_indentation.micaN.z;

		dr.w = 1.0f/dr.w;
		dr.w = dr.w*dr.w;
		r6 = dr.w*dr.w*dr.w;
		df.w = c_indentation.surfAprime*r6 + c_indentation.surfBprime;
		df.w = df.w*r6*dr.w;
		f.x += c_indentation.micaN.x*df.w + df.x;
		f.y += c_indentation.micaN.y*df.w + df.y;
		f.z += c_indentation.micaN.z*df.w + df.z;

		/*dr.w = powf(dr.w, 8);
		dr.w = 6.0*c_indentation.el*c_indentation.a6/dr.w;
		f.x += c_indentation.micaN.x*dr.w + df.x;
		f.y += c_indentation.micaN.y*dr.w + df.y;
		f.z += c_indentation.micaN.z*dr.w + df.z;*/
		c_gsop.d_forces[d_i] = f;
	}
}

__global__ void indentationDiscreteSurf_kernel(){
	int d_i = blockIdx.x*blockDim.x + threadIdx.x;
	if(d_i < c_gsop.aminoCount){
		float4 coord = c_gsop.d_coord[d_i];
		float4 f = c_gsop.d_forces[d_i];
		float4 dr;
		dr.x =  coord.x - c_indentation.tipCoord.x;
		dr.y =  coord.y - c_indentation.tipCoord.y;
		dr.z =  coord.z - c_indentation.tipCoord.z;
		dr.w = sqrtf(dr.x*dr.x + dr.y*dr.y + dr.z*dr.z);
		coord.w = dr.w - c_indentation.tipRadius;
		coord.w = 1.0f/coord.w;
		coord.w = coord.w*coord.w;
		float r6 = coord.w*coord.w*coord.w;
		coord.w = c_indentation.tipAprime*r6 + c_indentation.tipBprime;
		coord.w = coord.w*r6/(dr.w*(dr.w - c_indentation.tipRadius));
		dr.x = dr.x*coord.w;
		dr.y = dr.y*coord.w;
		dr.z = dr.z*coord.w;
		f.x += dr.x;
		f.y += dr.y;
		f.z += dr.z;

		coord = c_indentation.d_tipForces[d_i];
		coord.x += dr.x;
		coord.y += dr.y;
		coord.z += dr.z;
		c_indentation.d_tipForces[d_i] = coord;
		coord = c_gsop.d_coord[d_i];

		int i;
		for(i = 0; i < c_indentation.d_micaListCounts[d_i]; i++){
			dr = c_indentation.d_surfacePointsCoord[c_indentation.d_micaList[i*c_gsop.width + d_i]];

			dr.x = coord.x - dr.x;
			dr.y = coord.y - dr.y;
			dr.z = coord.z - dr.z;
			dr.w = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z;

			dr.w = 1.0f/dr.w;
			r6 = dr.w*dr.w*dr.w;
			coord.w = c_indentation.surfAprime*r6 + c_indentation.surfBprime;
			coord.w = coord.w*r6*dr.w;
			f.x += dr.x*coord.w;
			f.y += dr.y*coord.w;
			f.z += dr.z*coord.w;
		}

		c_gsop.d_forces[d_i] = f;
	}
}

__global__ void generateMicaList_kernel(){
	int d_i = blockIdx.x*blockDim.x + threadIdx.x;
	if(d_i < c_gsop.aminoCount){
		int i;
		int count = 0;
		float4 coord = c_gsop.d_coord[d_i];
		for(i = 0; i < c_indentation.surfaceBeadsCount; i++){
			float4 r2 = c_indentation.d_surfacePointsCoord[i];
			r2.x = coord.x - r2.x;
			r2.y = coord.y - r2.y;
			r2.z = coord.z - r2.z;
			r2.w = r2.x*r2.x + r2.y*r2.y + r2.z*r2.z;
			if(r2.w < c_indentation.pairsCutoff2){
				c_indentation.d_micaList[count*c_gsop.width + d_i] = i;
				count ++;
			}
		}
		c_indentation.d_micaListCounts[d_i] = count;
	}
}

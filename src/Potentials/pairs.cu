/*
 * pairs.cu
 *
 *  Created on: Jun 16, 2009
 *      Author: zhmurov
 */
#include "../gsop.cuh"
#include "pairs.h"


#include "pairs_kernel.cu"

void createPairsPotential(){
	potentials[potentialsCount] = new PairsPotential();
	potentialsCount++;
}

PairsPotential::PairsPotential(){
    this->name = "Long Range";
	printf("Initializing pairlist...");
	this->blockSize = getIntegerParameter(PAIRS_BLOCK_SIZE_STRING, gsop.blockSize, 1);
	this->blockNum = gsop.aminoCount/this->blockSize + 1;
	this->a = getFloatParameter(PAIRS_A_STRING, DEFAULT_PAIRS_A, 1);
	this->a2 = this->a*this->a;
	this->el = getFloatParameter(PAIRS_EL_STRING, DEFAULT_PAIRS_EL, 1);
	this->minus6elovera2 = -6.0f*this->el/this->a2;
	this->max_pairs = getIntegerParameter(MAX_PAIRS_STRING, DEFAULT_MAX_PAIRS, 1);
	this->pairsCutoff = getFloatParameter(PAIRS_CUTOFF_STRING, DEFAULT_PAIRS_CUTOFF, 1);
	this->pairsCutoff2 = this->pairsCutoff*this->pairsCutoff;

	// Allocating memory
	this->h_pairs = (int*)calloc(gsop.aminoCount*this->max_pairs, sizeof(int));
	this->h_pairsCount = (int*)calloc(gsop.aminoCount, sizeof(int));
	hipMalloc((void**)&this->d_pairs, gsop.aminoCount*this->max_pairs*sizeof(int));
	checkCUDAError();
	hipMalloc((void**)&this->d_pairsCount, gsop.aminoCount*sizeof(int));
    checkCUDAError();
    this->h_energies = (float*)calloc(gsop.aminoCount, sizeof(float));
    hipMalloc((void**)&this->d_energies, gsop.aminoCount*sizeof(float));
    this->energies = (float*)calloc(gsop.Ntr, sizeof(float));

    this->updateParametersOnGPU();
	printf("done.\n");

	if(deviceProp.major == 2){ // TODO: >= 2
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(pairs_kernel), hipFuncCachePreferL1);
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(pairsEnergy_kernel), hipFuncCachePreferL1);
	}
}

void PairsPotential::updateParametersOnGPU(){
    hc_pairs.pairsCutoff2 = this->pairsCutoff2;
    hc_pairs.a2 = this->a2;
    hc_pairs.el = this->el;
    hc_pairs.minus6elovera2 = this->minus6elovera2;
    hc_pairs.d_pairs = this->d_pairs;
    hc_pairs.d_pairsCount = this->d_pairsCount;
    hc_pairs.max_pairs = this->max_pairs;
    hc_pairs.d_energies = this->d_energies;
	hipMemcpyToSymbol(HIP_SYMBOL(c_pairs), &hc_pairs, sizeof(PairsConstant), 0, hipMemcpyHostToDevice);
	checkCUDAError();
}

void PairsPotential::compute(){
	pairs_kernel<<<this->blockNum, this->blockSize>>>();
	checkCUDAError();
}

int PairsPotential::getEnergiesCount(){
	return 1;
}

float* PairsPotential::computeEnergy(int id){
	if(id == 0){
		pairsEnergy_kernel<<<this->blockNum, this->blockSize>>>();
		hipMemcpy(this->h_energies, this->d_energies, gsop.aminoCount*sizeof(float), hipMemcpyDeviceToHost);
		SOPPotential::sumEnergies(this->h_energies, this->energies);
		checkCUDAError();
		return this->energies;
	} else {
		DIE("Pairs potential returns only one energy term");
		return NULL;
	}
}

float PairsPotential::getEnergy(int traj, int id){
	if(traj < gsop.Ntr && id == 0){
		return this->energies[traj];
	} else {
		DIE("Either trajectory or energy index is out of boundary");
		return 0.0f;
	}
}


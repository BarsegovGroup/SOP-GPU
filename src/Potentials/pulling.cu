#include "hip/hip_runtime.h"
/*
 * pulling.cu
 *
 *  Created on: May 25, 2010
 *      Author: zhmurov
 */
#include "../gsop.cuh"
#include "../IO/configreader.h"
#include "../Util/mystl.h"
#include "../Util/wrapper.h"
#include "../Util/vector_helpers.h"
#include "pulling.h"

#include "pulling_kernel.cu"

void createPullingPotential(){
	if(gsop.pullingOn == 1 || getYesNoParameter(PULLING_ON_STRING, 0, 1) == 1){
		gsop.pullingOn = 1;

        PullingPotential *pot;
		potentials[potentialsCount] = pot = new PullingPotential();
        potentialsCount++;

		if(getFloatParameter(PULLING_DELTAX_STRING, 0, 1) != 0){
			updaters[updatersCount] = new PullingUpdater(pot);
			updatersCount++;
		}
	}
}

PullingUpdater::PullingUpdater(PullingPotential *pulling){
	this->name = "Pulling";
    this->frequency = getIntegerParameter(PULLING_FREQ, gsop.nav, 1);
    this->pulling = pulling;
}

PullingPotential::PullingPotential(){
    this->name = "Pulling";
	printf("Initializing pulling protocol...\n");
	int i, j, traj;
	this->pullVector = (float3*)calloc(gsop.Ntr, sizeof(float3));
	this->cantCoord0 = (float3*)calloc(gsop.Ntr, sizeof(float3));
	this->cantCoord = (float3*)calloc(gsop.Ntr, sizeof(float3));
	this->extForce = (float3*)calloc(gsop.Ntr, sizeof(float3));
	this->h_extForces = (float4*)calloc(gsop.aminoCount, sizeof(float4));
	hipMalloc((void**)&this->d_extForces, gsop.aminoCount*sizeof(float4));

	this->deltax = getFloatParameter(PULLING_DELTAX_STRING, 0, 1);
	this->Ks = getFloatParameter(PULLING_KS_STRING, DEFAULT_PULLING_KS, 1);
	this->fconst = getFloatParameter(PULLING_FCONST_STRING, 0, 1);

	if(this->deltax == 0 && this->fconst == 0){
		DIE("ERROR: Either 'deltax' or 'fconst' parameter should be specified to initiate pulling\n");
	}

	this->fixedCount = getIntegerParameter(PULLING_FIXED_COUNT_STRING, 0, 0);
	this->fixed = (int*)malloc(this->fixedCount*sizeof(int));
	this->pulledCount = getIntegerParameter(PULLING_PULLED_COUNT_STRING, 0, 0);
	this->pulled = (int*)malloc(this->pulledCount*sizeof(int));
	printf("%d resid(s) fixed, %d pulled.\n", this->fixedCount, this->pulledCount);
	char paramName[10];
	for(i = 0; i < this->fixedCount; i++){
		sprintf(paramName, "%s%d", PULLING_FIXED_STRING, i+1);
		this->fixed[i] = getIntegerParameter(paramName, 0, 0);
		printf("Resid %d is fixed.\n", this->fixed[i]);
	}
	for(i = 0; i < this->pulledCount; i++){
		sprintf(paramName, "%s%d", PULLING_PULLED_STRING, i+1);
		this->pulled[i] = getIntegerParameter(paramName, 0, 0);
		printf("Pulling resid %d.\n", this->pulled[i]);
	}

	char pullDirection[30];
	getMaskedParameter(pullDirection, PULLING_DIRECTION_STRING, DEFAULT_PULLING_DIRECTION, 1);
	float3 pullVector;
	if(strcmp(pullDirection, PULLING_DIRECTION_VECTOR_STRING) == 0){
		this->fixedEnd = getIntegerParameter(PULLING_FIXED_END_STRING, 1, 1);
		this->pulledEnd = getIntegerParameter(PULLING_PULLED_END_STRING, 2, 1);
		getVectorParameter(PULLING_VECTOR_STRING, &pullVector.x, &pullVector.y, &pullVector.z, 0, 0, 0, 0);
		for(traj = 0; traj < gsop.Ntr; traj++){
			this->pullVector[traj] = pullVector;
			this->cantCoord0[traj] = make_float3(gsop.h_coord[traj*sop.aminoCount + this->pulledEnd]);
			this->cantCoord[traj]  = make_float3(gsop.h_coord[traj*sop.aminoCount + this->pulledEnd]);
		}
		printf("Pulling in direction of vector (%f, %f, %f).\n", pullVector.x, pullVector.y, pullVector.z);
	} else if(strcmp(pullDirection, PULLING_DIRECTION_ENDTOEND_STRING) == 0){
		this->fixedEnd = getIntegerParameter(PULLING_FIXED_END_STRING, 0, 0);
		this->pulledEnd = getIntegerParameter(PULLING_PULLED_END_STRING, 0, 0);
		for(traj = 0; traj < gsop.Ntr; traj++){
			this->pullVector[traj] = make_float3(
                    gsop.h_coord[traj*sop.aminoCount + this->pulledEnd]
					- gsop.h_coord[traj*sop.aminoCount + this->fixedEnd]);
			normalize(this->pullVector[traj]);
			this->cantCoord0[traj] = make_float3(gsop.h_coord[traj*sop.aminoCount + this->pulledEnd]);
			this->cantCoord[traj]  = make_float3(gsop.h_coord[traj*sop.aminoCount + this->pulledEnd]);
		}
		printf("Pulling in end-to-end direction: %c%d(%s) - %c%d(%s).\n",
				sop.aminos[this->fixedEnd].chain, sop.aminos[this->fixedEnd].resid, sop.aminos[this->fixedEnd].resName,
				sop.aminos[this->pulledEnd].chain, sop.aminos[this->pulledEnd].resid, sop.aminos[this->pulledEnd].resName);
		printf("Pulling vector is (%5.2f, %5.2f, %5.2f).\n",
				this->pullVector[0].x,
				this->pullVector[0].y,
				this->pullVector[0].z);
	} else {
		DIE("ERROR: 'pullDirection' parameter should be set to 'endToEnd' or 'vector'.\n");
	}

	for(traj = 0; traj < gsop.Ntr; traj++){
		for(i = 0; i < sop.aminoCount; i++){
			this->h_extForces[traj*sop.aminoCount + i] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
		}
		for(j = 0; j < this->pulledCount; j++){
			i = this->pulled[j];
			if(traj == 0){
				printf("Pulling bead #%d (%s %d chain %c).\n", i, sop.aminos[i].resName, sop.aminos[i].resid, sop.aminos[i].chain);
			}
			this->h_extForces[traj*sop.aminoCount + i] = make_float4(
					this->pullVector[traj] * this->fconst, 2.0f);
		}
		for(j = 0; j < this->fixedCount; j++){
			i = this->fixed[j];
			if(traj == 0){
				printf("Fixing bead #%d (%s %d chain %c).\n", i, sop.aminos[i].resName, sop.aminos[i].resid, sop.aminos[i].chain);
			}
			this->h_extForces[traj*sop.aminoCount + i] = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
		}
	}

	for(i = 0; i < sop.aminoCount; i++){
		sop.aminos[i].beta = this->h_extForces[i].w;
	}
	hipMemcpy(this->d_extForces, this->h_extForces, gsop.aminoCount*sizeof(float4), hipMemcpyHostToDevice);
    checkCUDAError();

    this->updateParametersOnGPU();

	if(this->deltax != 0.0f){
		pullFilenames.resize(gsop.Ntr);
        std::string pullFilename;
        pullFilename = getMaskedParameterAs<std::string>(PULLING_FILENAME, DEFAULT_PULLING_FILENAME);
		for(traj = 0; traj < gsop.Ntr; traj++){
			pullFilenames[traj] = string_replace(pullFilename, "<run>", traj+gsop.firstrun);
			FILE* pullFile = safe_fopen(pullFilenames[traj].c_str(), "w");
			fclose(pullFile);
		}
		printf("Pulling data will be saved in '%s'.\n", pullFilename.c_str());
	}
	
    this->blockSize = gsop.blockSize;
	this->blockNum = gsop.aminoCount/this->blockSize + 1;

	printf("Done initializing pulling protocol...\n");
}

void PullingPotential::compute(){
	// The force is added in integrator
}

void PullingPotential::updateForces(float xt){
	copyCoordDeviceToHost();
	int traj, j;
	for(traj = 0; traj < gsop.Ntr; traj++){
		this->extForce[traj] = this->computeForce(gsop.h_coord[sop.aminoCount*traj + this->pulledEnd], traj);
		// Increasing the force'
		this->cantCoord[traj] = this->cantCoord0[traj] + xt * this->pullVector[traj];
		for(j = 0; j < this->pulledCount; j++){
			this->h_extForces[traj*sop.aminoCount + this->pulled[j]] =
					make_float4(this->extForce[traj], 2.0f);
		}
	}
	hipMemcpy(this->d_extForces, this->h_extForces, gsop.aminoCount*sizeof(float4), hipMemcpyHostToDevice);
}

void PullingPotential::savePullingData(){
    int traj;
	if(gsop.step % 100000 == 0){
		printf("'Cantilever chip' coordinates for run #%d: %f, %f, %f\n",
				gsop.firstrun, this->cantCoord[0].x, this->cantCoord[0].y, this->cantCoord[0].z);
		printf("'Cantilever tip' coordinates for run #%d: %f, %f, %f\n",
				gsop.firstrun, gsop.h_coord[this->pulledEnd].x, gsop.h_coord[this->pulledEnd].y, gsop.h_coord[this->pulledEnd].z);
	}
	for(traj = 0; traj < gsop.Ntr; traj++){
		FILE* pullFile = safe_fopen(pullFilenames[traj].c_str(), "a");

        float3 endToEnd_vector = make_float3(gsop.h_coord[traj*sop.aminoCount + this->pulledEnd] - gsop.h_coord[traj*sop.aminoCount + this->fixedEnd]);
		float endToEnd_x = dot( endToEnd_vector , this->pullVector[traj] );
		float endToEnd = abs( endToEnd_vector );
        float f = dot(this->extForce[traj] , this->pullVector[traj]);

		fprintf(pullFile, "%12ld\t%5.3f\t%5.3f\t%5.3f\t%5.3f\t%5.3f\t%5.3f\n",
				gsop.step, endToEnd, endToEnd_x, f,
				this->extForce[traj].x, this->extForce[traj].y, this->extForce[traj].z);

		fclose(pullFile);
	}
	checkCUDAError();
}

void PullingPotential::updateParametersOnGPU(){
    hc_pulling.d_extForces = this->d_extForces;
	hipMemcpyToSymbol(HIP_SYMBOL(c_pulling), &hc_pulling, sizeof(PullingConstant), 0, hipMemcpyHostToDevice);
	checkCUDAError();
}

void PullingUpdater::update(){
    float xt = pulling->deltax*(gsop.step / this->frequency);
    pulling->updateForces(xt);
	if(gsop.step % this->frequency == 0){
        pulling->savePullingData();
	}
}

float3 PullingPotential::computeForce(const float4 &coordN, int traj) const{
	float3 f;

	f.x = this->Ks * (this->cantCoord[traj].x - coordN.x);
	f.y = this->Ks * (this->cantCoord[traj].y - coordN.y);
	f.z = this->Ks * (this->cantCoord[traj].z - coordN.z);

	return f;
}


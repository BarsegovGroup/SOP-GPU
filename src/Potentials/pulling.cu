#include "hip/hip_runtime.h"
/*
 * pulling.cu
 *
 *  Created on: May 25, 2010
 *      Author: zhmurov
 */
#include "../gsop.cuh"
#include "../IO/configreader.h"
#include "../Util/mystl.h"
#include "pulling.h"

#include "pulling_kernel.cu"

void createPullingPotential(){
	if(gsop.pullingOn == 1 || getYesNoParameter(PULLING_ON_STRING, 0, 1) == 1){
		gsop.pullingOn = 1;

        PullingPotential *pot;
		potentials[potentialsCount] = pot = new PullingPotential();
        potentialsCount++;

		if(getFloatParameter(PULLING_DELTAX_STRING, 0, 1) != 0){
			updaters[updatersCount] = new PullingUpdater(pot);
			updatersCount++;
		}
	}
}

PullingUpdater::PullingUpdater(PullingPotential *pulling){
	this->name = "Pulling";
    this->frequency = getIntegerParameter(PULLING_FREQ, nav, 1);
    this->pulling = pulling;
}

PullingPotential::PullingPotential(){
    this->name = "Pulling";
	printf("Initializing pulling protocol...\n");
	int i, j, traj;
	this->pullVector = (float3*)calloc(gsop.Ntr, sizeof(float3));
	this->cantCoord0 = (float3*)calloc(gsop.Ntr, sizeof(float3));
	this->cantCoord = (float3*)calloc(gsop.Ntr, sizeof(float3));
	this->extForce = (float3*)calloc(gsop.Ntr, sizeof(float3));
	this->h_extForces = (float4*)calloc(gsop.aminoCount, sizeof(float4));
	hipMalloc((void**)&this->d_extForces, gsop.aminoCount*sizeof(float4));

	this->deltax = getFloatParameter(PULLING_DELTAX_STRING, 0, 1);
	this->Ks = getFloatParameter(PULLING_KS_STRING, DEFAULT_PULLING_KS, 1);
	this->fconst = getFloatParameter(PULLING_FCONST_STRING, 0, 1);

	if(this->deltax == 0 && this->fconst == 0){
		DIE("ERROR: Either 'deltax' or 'fconst' parameter should be specified to initiate pulling\n");
	}

	this->fixedCount = getIntegerParameter(PULLING_FIXED_COUNT_STRING, 0, 0);
	this->fixed = (int*)malloc(this->fixedCount*sizeof(int));
	this->pulledCount = getIntegerParameter(PULLING_PULLED_COUNT_STRING, 0, 0);
	this->pulled = (int*)malloc(this->pulledCount*sizeof(int));
	printf("%d resid(s) fixed, %d pulled.\n", this->fixedCount, this->pulledCount);
	char paramName[10];
	for(i = 0; i < this->fixedCount; i++){
		sprintf(paramName, "%s%d", PULLING_FIXED_STRING, i+1);
		this->fixed[i] = getIntegerParameter(paramName, 0, 0);
		printf("Resid %d is fixed.\n", this->fixed[i]);
	}
	for(i = 0; i < this->pulledCount; i++){
		sprintf(paramName, "%s%d", PULLING_PULLED_STRING, i+1);
		this->pulled[i] = getIntegerParameter(paramName, 0, 0);
		printf("Pulling resid %d.\n", this->pulled[i]);
	}

	char pullDirection[30];
	getMaskedParameter(pullDirection, PULLING_DIRECTION_STRING, DEFAULT_PULLING_DIRECTION, 1);
	float3 pullVector;
	if(strcmp(pullDirection, PULLING_DIRECTION_VECTOR_STRING) == 0){
		this->fixedEnd = getIntegerParameter(PULLING_FIXED_END_STRING, 1, 1);
		this->pulledEnd = getIntegerParameter(PULLING_PULLED_END_STRING, 2, 1);
		getVectorParameter(PULLING_VECTOR_STRING, &pullVector.x, &pullVector.y, &pullVector.z, 0, 0, 0, 0);
		for(traj = 0; traj < gsop.Ntr; traj++){
			this->pullVector[traj].x = pullVector.x;
			this->pullVector[traj].y = pullVector.y;
			this->pullVector[traj].z = pullVector.z;
			this->cantCoord0[traj].x = gsop.h_coord[traj*sop.aminoCount + this->pulledEnd].x;
			this->cantCoord0[traj].y = gsop.h_coord[traj*sop.aminoCount + this->pulledEnd].y;
			this->cantCoord0[traj].z = gsop.h_coord[traj*sop.aminoCount + this->pulledEnd].z;
			this->cantCoord[traj].x = gsop.h_coord[traj*sop.aminoCount + this->pulledEnd].x;
			this->cantCoord[traj].y = gsop.h_coord[traj*sop.aminoCount + this->pulledEnd].y;
			this->cantCoord[traj].z = gsop.h_coord[traj*sop.aminoCount + this->pulledEnd].z;
		}
		printf("Pulling in direction of vector (%f, %f, %f).\n", pullVector.x, pullVector.y, pullVector.z);
	} else if(strcmp(pullDirection, PULLING_DIRECTION_ENDTOEND_STRING) == 0){
		this->fixedEnd = getIntegerParameter(PULLING_FIXED_END_STRING, 0, 0);
		this->pulledEnd = getIntegerParameter(PULLING_PULLED_END_STRING, 0, 0);
		for(traj = 0; traj < gsop.Ntr; traj++){
			this->pullVector[traj].x = gsop.h_coord[traj*sop.aminoCount + this->pulledEnd].x
					- gsop.h_coord[traj*sop.aminoCount + this->fixedEnd].x;
			this->pullVector[traj].y = gsop.h_coord[traj*sop.aminoCount + this->pulledEnd].y
					- gsop.h_coord[traj*sop.aminoCount + this->fixedEnd].y;
			this->pullVector[traj].z = gsop.h_coord[traj*sop.aminoCount + this->pulledEnd].z
					- gsop.h_coord[traj*sop.aminoCount + this->fixedEnd].z;
			float norm = sqrtf(this->pullVector[traj].x*this->pullVector[traj].x +
					this->pullVector[traj].y*this->pullVector[traj].y +
					this->pullVector[traj].z*this->pullVector[traj].z);
			this->pullVector[traj].x /= norm;
			this->pullVector[traj].y /= norm;
			this->pullVector[traj].z /= norm;
			this->cantCoord0[traj].x = gsop.h_coord[traj*sop.aminoCount + this->pulledEnd].x;
			this->cantCoord0[traj].y = gsop.h_coord[traj*sop.aminoCount + this->pulledEnd].y;
			this->cantCoord0[traj].z = gsop.h_coord[traj*sop.aminoCount + this->pulledEnd].z;
			this->cantCoord[traj].x = gsop.h_coord[traj*sop.aminoCount + this->pulledEnd].x;
			this->cantCoord[traj].y = gsop.h_coord[traj*sop.aminoCount + this->pulledEnd].y;
			this->cantCoord[traj].z = gsop.h_coord[traj*sop.aminoCount + this->pulledEnd].z;
		}
		printf("Pulling in end-to-end direction: %c%d(%s) - %c%d(%s).\n",
				sop.aminos[this->fixedEnd].chain, sop.aminos[this->fixedEnd].resid, sop.aminos[this->fixedEnd].resName,
				sop.aminos[this->pulledEnd].chain, sop.aminos[this->pulledEnd].resid, sop.aminos[this->pulledEnd].resName);
		printf("Pulling vector is (%5.2f, %5.2f, %5.2f).\n",
				this->pullVector[0].x,
				this->pullVector[0].y,
				this->pullVector[0].z);
	} else {
		DIE("ERROR: 'pullDirection' parameter should be set to 'endToEnd' or 'vector'.\n");
	}

	for(traj = 0; traj < gsop.Ntr; traj++){
		for(i = 0; i < sop.aminoCount; i++){
			this->h_extForces[traj*sop.aminoCount + i] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
		}
		for(j = 0; j < this->pulledCount; j++){
			i = this->pulled[j];
			if(traj == 0){
				printf("Pulling bead #%d (%s %d chain %c).\n", i, sop.aminos[i].resName, sop.aminos[i].resid, sop.aminos[i].chain);
			}
			this->h_extForces[traj*sop.aminoCount + i] = make_float4(
					this->pullVector[traj].x*this->fconst,
					this->pullVector[traj].y*this->fconst,
					this->pullVector[traj].z*this->fconst, 2.0);
		}
		for(j = 0; j < this->fixedCount; j++){
			i = this->fixed[j];
			if(traj == 0){
				printf("Fixing bead #%d (%s %d chain %c).\n", i, sop.aminos[i].resName, sop.aminos[i].resid, sop.aminos[i].chain);
			}
			this->h_extForces[traj*sop.aminoCount + i] = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
		}
	}

	for(i = 0; i < sop.aminoCount; i++){
		sop.aminos[i].beta = this->h_extForces[i].w;
	}
	hipMemcpy(this->d_extForces, this->h_extForces, gsop.aminoCount*sizeof(float4), hipMemcpyHostToDevice);
    checkCUDAError();

    hc_pulling.d_extForces = this->d_extForces;
	hipMemcpyToSymbol(HIP_SYMBOL(c_pulling), &hc_pulling, sizeof(PullingConstant), 0, hipMemcpyHostToDevice);
	checkCUDAError();

	if(this->deltax != 0.0f){
		pullFilenames.resize(gsop.Ntr);
        std::string pullFilename;
        pullFilename = getMaskedParameterAs<std::string>(PULLING_FILENAME, DEFAULT_PULLING_FILENAME);
		for(traj = 0; traj < gsop.Ntr; traj++){
			pullFilenames[traj] = string_replace(pullFilename, "<run>", traj+gsop.firstrun);
			FILE* pullFile = fopen(pullFilenames[traj].c_str(), "w");
			fclose(pullFile);
		}
		printf("Pulling data will be saved in '%s'.\n", pullFilename.c_str());
	}
	
    this->blockSize = getIntegerParameter(COVALENT_BLOCK_SIZE_STRING, gsop.blockSize, 1);
	this->blockNum = gsop.aminoCount/this->blockSize + 1;

	printf("Done initializing pulling protocol...\n");
}

void PullingPotential::compute(){
	pulling_kernel<<<this->blockNum, this->blockSize>>>();
	checkCUDAError();
}

void PullingPotential::computeEnergy(){

}

void PullingPotential::updateForces(float xt){
	copyCoordDeviceToHost();
	int traj, j;
	for(traj = 0; traj < gsop.Ntr; traj++){
		this->extForce[traj] = this->computeForce(gsop.h_coord[sop.aminoCount*traj + this->pulledEnd], traj);
		// Increasing the force'
		this->cantCoord[traj].x = this->cantCoord0[traj].x + xt * this->pullVector[traj].x;
		this->cantCoord[traj].y = this->cantCoord0[traj].y + xt * this->pullVector[traj].y;
		this->cantCoord[traj].z = this->cantCoord0[traj].z + xt * this->pullVector[traj].z;
		for(j = 0; j < this->pulledCount; j++){
			this->h_extForces[traj*sop.aminoCount + this->pulled[j]] =
					make_float4(this->extForce[traj].x, this->extForce[traj].y, this->extForce[traj].z, 2.0);
		}
	}
	hipMemcpy(this->d_extForces, this->h_extForces, gsop.aminoCount*sizeof(float4), hipMemcpyHostToDevice);
}

void PullingPotential::savePullingData(){
    int traj;
	if(step % 100000 == 0){
		printf("'Cantilever chip' coordinates for run #%d: %f, %f, %f\n",
				gsop.firstrun, this->cantCoord[0].x, this->cantCoord[0].y, this->cantCoord[0].z);
		printf("'Cantilever tip' coordinates for run #%d: %f, %f, %f\n",
				gsop.firstrun, gsop.h_coord[this->pulledEnd].x, gsop.h_coord[this->pulledEnd].y, gsop.h_coord[this->pulledEnd].z);
	}
	for(traj = 0; traj < gsop.Ntr; traj++){
		FILE* pullFile = fopen(pullFilenames[traj].c_str(), "a");

		float endToEnd_x = (gsop.h_coord[traj*sop.aminoCount + this->pulledEnd].x - gsop.h_coord[traj*sop.aminoCount + this->fixedEnd].x)*this->pullVector[traj].x +
				(gsop.h_coord[traj*sop.aminoCount + this->pulledEnd].y - gsop.h_coord[traj*sop.aminoCount + this->fixedEnd].y)*this->pullVector[traj].y +
				(gsop.h_coord[traj*sop.aminoCount + this->pulledEnd].z - gsop.h_coord[traj*sop.aminoCount + this->fixedEnd].z)*this->pullVector[traj].z;
		float endToEnd = sqrtf((gsop.h_coord[traj*sop.aminoCount + this->pulledEnd].x-gsop.h_coord[traj*sop.aminoCount + this->fixedEnd].x)*
				(gsop.h_coord[traj*sop.aminoCount + this->pulledEnd].x-gsop.h_coord[traj*sop.aminoCount + this->fixedEnd].x) +
				(gsop.h_coord[traj*sop.aminoCount + this->pulledEnd].y-gsop.h_coord[traj*sop.aminoCount + this->fixedEnd].y)*
				(gsop.h_coord[traj*sop.aminoCount + this->pulledEnd].y-gsop.h_coord[traj*sop.aminoCount + this->fixedEnd].y) +
				(gsop.h_coord[traj*sop.aminoCount + this->pulledEnd].z-gsop.h_coord[traj*sop.aminoCount + this->fixedEnd].z)*
				(gsop.h_coord[traj*sop.aminoCount + this->pulledEnd].z-gsop.h_coord[traj*sop.aminoCount + this->fixedEnd].z));
		float f = this->extForce[traj].x*this->pullVector[traj].x + this->extForce[traj].y*this->pullVector[traj].y + this->extForce[traj].z*this->pullVector[traj].z;

		fprintf(pullFile, "%12ld\t%5.3f\t%5.3f\t%5.3f\t%5.3f\t%5.3f\t%5.3f\n",
				step, endToEnd, endToEnd_x, f,
				this->extForce[traj].x, this->extForce[traj].y, this->extForce[traj].z);

		fclose(pullFile);
	}
	checkCUDAError();
}

void PullingUpdater::update(){
    float xt = pulling->deltax*(step / this->frequency);
    pulling->updateForces(xt);
	if(step % this->frequency == 0){
        pulling->savePullingData();
	}
}

float3 PullingPotential::computeForce(float4 coordN, int traj) const{
	float3 f;

	f.x = this->Ks * (this->cantCoord[traj].x - coordN.x);
	f.y = this->Ks * (this->cantCoord[traj].y - coordN.y);
	f.z = this->Ks * (this->cantCoord[traj].z - coordN.z);

	return f;
}


#include "hip/hip_runtime.h"
/*
 * pulling_plane.cu
 *
 *  Created on: Jan 17, 2014
 *      Author: alekseenko
 */
#include "../gsop.cuh"
#include "../Integrators/langevin.h"
#include "../IO/configreader.h"
#include "pulling_plane.h"

char pullingPlaneFilename[500];
FILE* pullingPlaneFile;

#include "pulling_plane_kernel.cu"

void createPullingPlanePotential(){
	if(getYesNoParameter(PULLINGPLANE_ON_STRING, 0, 1) == 1){
		gsop.pullingPlaneOn = 1;

        PullingPlanePotential *pot;
		potentials[potentialsCount] = pot = new PullingPlanePotential();
		potentialsCount++;

		updaters[updatersCount] = new PullingPlaneUpdater(pot);
		updatersCount++;
	}
}

PullingPlaneUpdater::PullingPlaneUpdater(PullingPlanePotential *pullingPlane){
	this->name = "Pulling Plane";
    this->frequency = getIntegerParameter(PULLINGPLANE_FREQ, nav, 1);
    this->pullingPlane = pullingPlane;
}

PullingPlanePotential::PullingPlanePotential(){
    this->name = "Pulling Plane";
	printf("Initializing pulling plane protocol...\n");
    if (gsop.Ntr != 1) {
        DIE("Pulling plane can only run in single-trajectory-per-GPU mode (runnum 1)\n");
    }
	int i, j;
	this->h_extForces = (float4*)calloc(gsop.aminoCount, sizeof(float4));
	hipMalloc((void**)&this->d_extForces, gsop.aminoCount*sizeof(float4));

	this->deltax = getFloatParameter(PULLINGPLANE_DELTAX_STRING, 0, 1);
	this->Ks = getFloatParameter(PULLINGPLANE_KS_STRING, DEFAULT_PULLINGPLANE_KS, 1);

	if(this->deltax == 0){
		printf("ERROR: '%s' parameter should be specified to initiate this->\n", PULLINGPLANE_DELTAX_STRING);
		exit(-1);
	}

	this->fixedCount = getIntegerParameter(PULLINGPLANE_FIXED_COUNT_STRING, 0, 0);
	this->fixed = (int*)malloc(this->fixedCount*sizeof(int));
	this->pulledCount = getIntegerParameter(PULLINGPLANE_PULLED_COUNT_STRING, 0, 0);
	this->pulled = (int*)malloc(this->pulledCount*sizeof(int));
	printf("%d resid(s) fixed, %d pulled.\n", this->fixedCount, this->pulledCount);
	char paramName[50];
	for(i = 0; i < this->fixedCount; i++){
		sprintf(paramName, "%s%d", PULLINGPLANE_FIXED_STRING, i+1);
		this->fixed[i] = getIntegerParameter(paramName, 0, 0);
		if(this->fixed[i] < 0 || this->fixed[i] >= gsop.aminoCount){
			DIE("ERROR: Fixed bead %s %d not exists. Protein has only %d amino-acids. Bead numbers should start with zero.\n", paramName, this->fixed[i], gsop.aminoCount);
		}
		printf("Resid %d is fixed.\n", this->fixed[i]);
	}
	for(i = 0; i < this->pulledCount; i++){
		sprintf(paramName, "%s%d", PULLINGPLANE_PULLED_STRING, i+1);
		this->pulled[i] = getIntegerParameter(paramName, 0, 0);
		if(this->pulled[i] < 0 || this->pulled[i] >= gsop.aminoCount){
			DIE("ERROR: Pulled bead %s %d not exists. Protein has only %d amino-acids. Bead numbers should start with zero.\n", paramName, this->pulled[i], gsop.aminoCount);
		}
		printf("Pulling resid %d.\n", this->pulled[i]);
	}

    getVectorParameter(PULLINGPLANE_PULLVECTOR_STRING, &this->pullVector.x, &this->pullVector.y, &this->pullVector.z, 0, 0, 0, 0);
    double t = sqrt(this->pullVector.x*this->pullVector.x + this->pullVector.y*this->pullVector.y + this->pullVector.z*this->pullVector.z);
    this->pullVector.x /= t;
    this->pullVector.y /= t;
    this->pullVector.z /= t;
    getVectorParameter(PULLINGPLANE_ZEROVECTOR_STRING, &this->planeCoord0.x, &this->planeCoord0.y, &this->planeCoord0.z, 0, 0, 0, 0);
    this->d = - (this->planeCoord.x*this->pullVector.x + this->planeCoord.y*this->pullVector.y + this->planeCoord.z*this->pullVector.z);
    this->cant_d = this->d;

	for(i = 0; i < sop.aminoCount; i++){
		this->h_extForces[i] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	}
	for(j = 0; j < this->pulledCount; j++){
		i = this->pulled[j];
		printf("Pulling bead #%d (%s %d chain %c).\n", i, sop.aminos[i].resName, sop.aminos[i].resid, sop.aminos[i].chain);
		this->h_extForces[i] = make_float4(0.0, 0.0, 0.0, 2.0);
	}
	for(j = 0; j < this->fixedCount; j++){
		i = this->fixed[j];
		printf("Fixing bead #%d (%s %d chain %c).\n", i, sop.aminos[i].resName, sop.aminos[i].resid, sop.aminos[i].chain);
		this->h_extForces[i] = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
	}

	for(i = 0; i < sop.aminoCount; i++){
		sop.aminos[i].beta = this->h_extForces[i].w;
	}
	hipMemcpy(this->d_extForces, this->h_extForces, gsop.aminoCount*sizeof(float4), hipMemcpyHostToDevice);
    checkCUDAError();

    hc_pullingPlane.d_extForces = this->d_extForces;
    hc_pullingPlane.pullVector = this->pullVector;
    hc_pullingPlane.d = this->d;
    hc_pullingPlane.Ks = this->Ks;
	hipMemcpyToSymbol(HIP_SYMBOL(c_pullingPlane), &hc_pullingPlane, sizeof(PullingPlaneConstant), 0, hipMemcpyHostToDevice);
	checkCUDAError();

    char tmpstr[512];
	getMaskedParameter(tmpstr, PULLINGPLANE_FILENAME, "", 0);
	char trajnum[10];
	sprintf(trajnum, "%d", gsop.firstrun);
	replaceString(pullingPlaneFilename, tmpstr, trajnum, "<run>");
	pullingPlaneFile = fopen(pullingPlaneFilename, "w");
	fclose(pullingPlaneFile);
	printf("PullingPlane data will be saved in '%s'.\n", pullingPlaneFilename);
	
    if(deviceProp.major == 2){ // TODO: >= 2
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(pullingPlane_kernel), hipFuncCachePreferL1);
	}

    this->blockSize = getIntegerParameter(COVALENT_BLOCK_SIZE_STRING, gsop.blockSize, 1);
	this->blockNum = gsop.aminoCount/this->blockSize + 1;

	printf("Done initializing pulling plane protocol...\n");
}

void PullingPlanePotential::compute(){
	pullingPlane_kernel<<<this->blockNum, this->blockSize>>>();
	checkCUDAError();
}

void PullingPlanePotential::computeEnergy(){
    // Perl has operator "..."
    // It should be used inside functions to be implemented
    // Unlike just empty functions, the warning is generated when "..." is used
    // This comment is completely useless and barely related, because this function is empty by design
}

void PullingPlaneUpdater::update(){

	//copyCoordDeviceToHost();
	int j;
	// Increasing the force
	float xt = pullingPlane->deltax * step / this->frequency;
    /*
	pullingPlane->cantCoord.x = pullingPlane->planeCoord0.x + xt * pullingPlane->pullVector.x;
	pullingPlane->cantCoord.y = pullingPlane->planeCoord0.y + xt * pullingPlane->pullVector.y;
	pullingPlane->cantCoord.z = pullingPlane->planeCoord0.z + xt * pullingPlane->pullVector.z;
    */
    pullingPlane->cant_d = pullingPlane->d0 + xt;
	checkCUDAError();
	hipMemcpy(pullingPlane->h_extForces, pullingPlane->d_extForces, gsop.aminoCount*sizeof(float4), hipMemcpyDeviceToHost);
	checkCUDAError();
	if(step % this->frequency == 0){
		if(step % 100000 == 0){
			printf("Cantilever coordinates for run #%d: %f, %f, %f\n",
					gsop.firstrun, pullingPlane->cantCoord.x, pullingPlane->cantCoord.y, pullingPlane->cantCoord.z);
			printf("Plane coordinates for run #%d: %f, %f, %f\n",
					gsop.firstrun, pullingPlane->planeCoord.x, pullingPlane->planeCoord.y, pullingPlane->planeCoord.z);
		}
        pullingPlaneFile = fopen(pullingPlaneFilename, "a");
        float3 extForce = make_float3(0.f, 0.f, 0.f);
        for (j = 0; j < gsop.aminoCount; j++){
            extForce.x += pullingPlane->h_extForces[j].x;
            extForce.y += pullingPlane->h_extForces[j].y;
            extForce.z += pullingPlane->h_extForces[j].z;
        }

        float extForceProj = extForce.x*pullingPlane->pullVector.x + extForce.y*pullingPlane->pullVector.y + extForce.z*pullingPlane->pullVector.z;

        float totForce = pullingPlane->Ks * (pullingPlane->cant_d - pullingPlane->d) - extForceProj;

        pullingPlane->d += totForce * this->frequency * ((LangevinIntegrator*)integrator)->hOverZeta; // TODO: fix dependency

        hc_pullingPlane.d = pullingPlane->d;
        hipMemcpyToSymbol(HIP_SYMBOL(c_pullingPlane), &hc_pullingPlane, sizeof(PullingPlaneConstant), 0, hipMemcpyHostToDevice);

        fprintf(pullingPlaneFile, "%12ld\t%5.3f\t%5.3f\t%5.3f\t%5.3f\t%5.3f\t%5.3f\n",
                step, pullingPlane->d, pullingPlane->cant_d, extForceProj,
                extForce.x, extForce.y, extForce.z);

        fflush(pullingPlaneFile);
        fclose(pullingPlaneFile);
    }

	checkCUDAError();
}


#include "hip/hip_runtime.h"
/*
 * native.cu
 *
 *  Created on: Jun 16, 2009
 *      Author: zhmurov
 */
#include "../gsop.cuh"
#include "../Util/wrapper.h"
#include "native.h"
//#define DEBUG1

#include "native_kernel.cu"

void createNativePotential(){
	potentials[potentialsCount] = new NativePotential();
	potentialsCount++;
}

NativePotential::NativePotential(){
	this->name = "Native";

	printf("Building map of native contacts...\n");

	// Reading parameters
	this->R_limit_bond = parameters::R_limit_bond.get();
	this->max_native = parameters::max_native.get();
	this->blockSize = gsop.blockSize;
	this->blockNum = gsop.aminoCount/this->blockSize + 1;

	// Allocating memory
	this->h_native = (int*)calloc(gsop.aminoCount*this->max_native, sizeof(int));
	hipMalloc((void**)&this->d_native, gsop.aminoCount*this->max_native*sizeof(int));
	this->h_nativeCount = (int*)calloc(gsop.aminoCount, sizeof(int));
	hipMalloc((void**)&this->d_nativeCount, gsop.aminoCount*sizeof(int));
	this->h_nativeParameters = (GNativeParameters*)calloc(this->max_native*gsop.aminoCount, sizeof(GNativeParameters));
	hipMalloc((void**)&this->d_nativeParameters, this->max_native*gsop.aminoCount*sizeof(GNativeParameters));
	this->h_energies = (float*)calloc(gsop.aminoCount, sizeof(float));
	hipMalloc((void**)&this->d_energies, gsop.aminoCount*sizeof(float));
	this->energies = (float*)calloc(gsop.Ntr, sizeof(float));

	// Building the native contacts map
    this->buildMap();

	// Copying to device
	hipMemcpy(this->d_native, this->h_native, this->max_native*gsop.aminoCount*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(this->d_nativeCount, this->h_nativeCount, gsop.aminoCount*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(this->d_nativeParameters, this->h_nativeParameters, this->max_native*gsop.aminoCount*sizeof(float2), hipMemcpyHostToDevice);

    this->updateParametersOnGPU();
	printf("Total number of native contacts: %d \n", totalNative);

	if(deviceProp.major == 2){ // TODO: >= 2
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(native_kernel), hipFuncCachePreferL1);
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(nativeEnergy_kernel), hipFuncCachePreferL1);
	}
}

void NativePotential::buildMap(){
	// Building map
    totalNative = 0;
	int i, j, k;
	for(k = 0; k < sop.nativeCount; k++){
		i = sop.natives[k].i;
		j = sop.natives[k].j;
		this->h_native[this->h_nativeCount[i]*gsop.aminoCount + i] = j;
		this->h_native[this->h_nativeCount[j]*gsop.aminoCount + j] = i;
		this->h_nativeParameters[this->h_nativeCount[i]*gsop.aminoCount + i].r02 = sop.natives[k].r0*sop.natives[k].r0;
		this->h_nativeParameters[this->h_nativeCount[j]*gsop.aminoCount + j].r02 = sop.natives[k].r0*sop.natives[k].r0;
		this->h_nativeParameters[this->h_nativeCount[i]*gsop.aminoCount + i].minus12ehOverR02 =
														-12.0*sop.natives[k].eh/(sop.natives[k].r0*sop.natives[k].r0);
		this->h_nativeParameters[this->h_nativeCount[j]*gsop.aminoCount + j].minus12ehOverR02 =
														-12.0*sop.natives[k].eh/(sop.natives[k].r0*sop.natives[k].r0);
		this->h_nativeCount[i] ++;
		this->h_nativeCount[j] ++;
		if(this->h_nativeCount[i] > this->max_native || this->h_nativeCount[j] > this->max_native){
			DIE("ERROR: Maximum number of native contacts exceeded the limit of %d.", this->max_native);
		}
		totalNative++;
	}

	// Multiply map over trajectories for many-runs-per-GPU
	for(j = 1; j < gsop.Ntr; j++){
		for(i = 0; i < sop.aminoCount; i++){
			for(k = 0; k < this->max_native; k++){
				this->h_native[j*sop.aminoCount + i + k*gsop.aminoCount] =
							this->h_native[i + k*gsop.aminoCount] + j*sop.aminoCount;
				this->h_nativeParameters[j*sop.aminoCount + i + k*gsop.aminoCount] =
							this->h_nativeParameters[i + k*gsop.aminoCount];
			}
			this->h_nativeCount[j*sop.aminoCount + i] = this->h_nativeCount[i];
		}
	}


	#ifdef DEBUG1
	printf("Native contacts (number of contacts, #s of beads, zeros):\n");
	for(int i = 0; i < gsop.aminoCount; i++){
		printf("%d (%d): ", i, this->h_nativeCount[i]);
		for(int j = 0; j < this->h_nativeCount[i]; j++){
			printf("%d(%3.1f,%3.1f)  ", this->h_native[j*gsop.aminoCount + i],
					this->h_nativeParameters[j*gsop.aminoCount + i].x,
					this->h_nativeParameters[j*gsop.aminoCount + i].y);
		}
		printf("\n");
	}
	#endif
}

void NativePotential::updateParametersOnGPU(){
    hc_native.d_native = this->d_native;
    hc_native.d_nativeCount = this->d_nativeCount;
    hc_native.d_nativeParameters = this->d_nativeParameters;
    hc_native.d_energies = this->d_energies;
	hipMemcpyToSymbol(HIP_SYMBOL(c_native), &hc_native, sizeof(NativeConstant), 0, hipMemcpyHostToDevice);
    checkCUDAError();
}

void NativePotential::compute(){
	native_kernel<<<this->blockNum, this->blockSize>>>();
	checkCUDAError();
}

int NativePotential::getEnergiesCount(){
	return 1;
}

float* NativePotential::computeEnergy(int id){
	if(id == 0){
		nativeEnergy_kernel<<<this->blockNum, this->blockSize>>>();
		hipMemcpy(this->h_energies, this->d_energies, gsop.aminoCount*sizeof(float), hipMemcpyDeviceToHost);
		SOPPotential::sumEnergies(this->h_energies, this->energies);
		checkCUDAError();
		return this->energies;
	} else {
		DIE("Native potential can compute only one energy term.\n");
		return NULL;
	}
}

float NativePotential::getEnergy(int traj, int id){
	if(traj < gsop.Ntr && id == 0){
		return this->energies[traj];
	} else {
		DIE("Either trajectory or energy index is out of bounds");
		return 0.0f;
	}
}


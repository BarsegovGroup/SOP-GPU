#include "hip/hip_runtime.h"
/*
 * gsop.cu
 *
 *  Created on: Jun 4, 2009
 *      Author: zhmurov
 */
#include "def_param.h"
#include "gsop.cuh"
#include "Potentials/covalent.cu"
#include "Potentials/native.cu"
#include "Potentials/pairs.cu"
#include "Potentials/indentation.cu"
#include "Potentials/pulling.cu"
#include "Updaters/pairlist.cu"
#include "Updaters/possiblepairlist.cu"
#include "Updaters/output_manager.cu"
#include "Updaters/dcd_manager.cu"
#include "Integrators/langevin.cu"
//#include "externalForce.cu"

void initGPU();
void initFF();
void runGPU();
void bindTextures();
void checkCUDAError();

extern void savePDB(char* pdb_filename);

void initCoordinates();
void copyCoordinates();
void copyCoordinatesTrajectory(int traj);
void initForces();
void initEnergies();

extern void replaceString(char* resultString, const char* initialString, const char* replacementString, const char* stringToReplace);
extern void initTemperature();
extern void initCovalent();
extern void initNative();
extern void initPairs();
extern uint4* initRandom(int seed, int N);
extern void shiftRandomSeeds(uint4* d_newseeds, int N);

unsigned int cpuTimer;
double gpuTime = 0.0;
double cpuTime = 0.0;
extern double pairlistTime = 0.0;
extern double covalentTime = 0.0;
extern double nativeTime = 0.0;
extern double pairsTime = 0.0;

long int lastStepCoordCopied = -1;

GSOP gsop;
SOPPotential** potentials;
SOPUpdater** updaters;
SOPIntegrator* integrator;

int potentialsCount;
int updatersCount;

/*
 * Prepare data on Host, copy it into Device, bind textures
 */
void initGPU(){

	//initDCD();
	hipSetDevice(device);
	hipGetDeviceProperties(&gsop.deviceProp, device);
	printf("Using device %d: \"%s\"\n", device, gsop.deviceProp.name);
	printf("CUDA Capability revision number: %d.%d\n", gsop.deviceProp.major, gsop.deviceProp.minor);
	gsop.aminoCount = Ntr*sop.aminoCount;
	gsop.width = gsop.aminoCount;
	while(gsop.width % 8 != 0){
		gsop.width++;
	}
	printf("Will align structures to width of %d.\n", gsop.width);

	initCoordinates(); // Alocate memory for coordinates
	initForces(); // Allocate memory for forces
}

void initFF(){

	potentialsCount = 0;
	updatersCount = 0;
	gsop.minimizationOn = minimizationOn;
	gsop.pullingOn = pullingOn;

	// Allocating memory for the model
	int i;
	potentials = (SOPPotential**)calloc(max_potentials, sizeof(SOPPotential*));
	for(i = 0; i < max_potentials; i++){
		potentials[i] = (SOPPotential*)malloc(sizeof(SOPPotential));
	}
	updaters = (SOPUpdater**)calloc(max_updaters, sizeof(SOPUpdater*));
	for(i = 0; i < max_updaters; i++){
		updaters[i] = (SOPUpdater*)malloc(sizeof(SOPUpdater));
	}
	integrator = (SOPIntegrator*)malloc(sizeof(SOPIntegrator));

	// Creating model
	createCovalentPotential(); // FENE
	createNativePotential(); // Full LJ
	createPairsPotential(); // Repulsive LJ
	createIndentationPotential(); // Indentations sphere and surface
	createPullingPotential(); // External force

	if(Ntr == 1){
		createPossiblepairlistUpdater(); // Updates the list of all pairs (for Verlet list)
	}
	createOutputManager(); // Save dat output
	createPairlistUpdater(); // Verlet list
	createDCDOutputManager(); // Save coordinates (dcd + pdb restart)

	createLangevinIntegrator(); // Create integrator

	initEnergies(); // Allocate memory for energy output (move to initGPU() ?)

	hipMemcpyToSymbol(HIP_SYMBOL(c_gsop), &gsop, sizeof(GSOP), 0, hipMemcpyHostToDevice);
	bindTextures();
	checkCUDAError();
}

/*
 *
 */
void runGPU(){
	printf("Starting simulations.\n");
	/*if(stage == pull_stage){
		engageCantileverTip();
	}*/

	int i, j, p, u;

	// Leave internal loop only when updater execution is needed (most frequent updater)
	int stride = nav;
	for(u = 0; u < updatersCount; u ++){
		if(updaters[u]->frequency < stride){
			stride = updaters[u]->frequency;
		}
	}
	int size = gsop.aminoCount*sizeof(float4);
	generatePairlist();

	// External loop
	for(i = step/stride; i <= numsteps/stride; i++){
		// Run all updaters
		for(u = 0; u < updatersCount; u++){
			updaters[u]->update();
		}
		// Internal loop
		for(j = 0; j < stride; j++){
			// Compute all potentials
			for(p = 0; p < potentialsCount; p++){
				potentials[p]->compute();
			}
			integrator->integrate(); // Integrate
			checkCUDAError();
		}

		//hipMemcpy(gsop.h_coord, gsop.d_coord, size, hipMemcpyDeviceToHost);
		step += stride;

		checkCUDAError();
	}
	copyCoordDeviceToHost();


	int traj;
	for(traj = 0; traj < Ntr; traj++){
		char trajnum[10];
		char trajCoordFilename[100];
		sprintf(trajnum, "%d\0", traj+firstrun);
		replaceString(trajCoordFilename, final_filename, trajnum, "<run>");
		for(i = 0; i < sop.aminoCount; i++){
			sop.aminos[i].x = gsop.h_coord[sop.aminoCount*traj + i].x;
			sop.aminos[i].y = gsop.h_coord[sop.aminoCount*traj + i].y;
			sop.aminos[i].z = gsop.h_coord[sop.aminoCount*traj + i].z;
		}
		savePDB(trajCoordFilename);
	}

}

void initCoordinates(){
	printf("Initializing coordinates (%d particles)...\n", gsop.aminoCount);
	int size = gsop.aminoCount*sizeof(float4);
	//gsop.h_coord = (float4*)calloc(gsop.aminoCount, sizeof(float4));
	hipHostMalloc((void**)&gsop.h_coord, size);
	hipMalloc((void**)&gsop.d_coord, size);
	//hipMalloc((void**)&gsop.d_coordToSave, size);
}

void copyCoordinates(){
	printf("Copying coordinates...\n");
	int size = gsop.aminoCount*sizeof(float4);
	hipMemcpy(gsop.d_coord, gsop.h_coord, size, hipMemcpyHostToDevice);
	//hipMemcpy(gsop.d_coordToSave, gsop.h_coord, size, hipMemcpyHostToDevice);
}

void copyCoordDeviceToHost(){
	if(step != lastStepCoordCopied){
		hipMemcpy(gsop.h_coord, gsop.d_coord, gsop.aminoCount*sizeof(float4), hipMemcpyDeviceToHost);
		lastStepCoordCopied = step;
	}
}

void copyCoordinatesTrajectory(int traj){
	int i;
	for(i = 0; i < sop.aminoCount; i++){
		gsop.h_coord[traj*sop.aminoCount + i].x = sop.aminos[i].x;
		gsop.h_coord[traj*sop.aminoCount + i].y = sop.aminos[i].y;
		gsop.h_coord[traj*sop.aminoCount + i].z = sop.aminos[i].z;
		gsop.h_coord[traj*sop.aminoCount + i].w = 0;
	}
#ifdef DEBUG
	printf("Coordinates for run #%d:\n", traj+firstrun);
	for(i = 0; i < sop.aminoCount; i++){
		printf("%d:\t%f\t%f\t%f\n", i, sop.aminos[i].x, sop.aminos[i].y, sop.aminos[i].z);
	}
#endif
}

/*
 * Initialize array of forces
 */
void initForces(){
	// Allocating memory
	printf("Allocating memory for forces...\n");
	int size = gsop.aminoCount*sizeof(float4);
	gsop.h_forces = (float4*)calloc(gsop.aminoCount, sizeof(float4));
	hipMalloc((void**)&gsop.d_forces, size);
	// Copying to the Device
	hipMemcpy(gsop.d_forces, gsop.h_forces, size, hipMemcpyHostToDevice);
}

/*
 * Initialize array of energies for output
 */
void initEnergies(){
	// Allocating memory
	printf("Allocating memory for energies...\n");
	int size = gsop.aminoCount*sizeof(float4);
	//gsop.h_energies = (float4*)calloc(gsop.aminoCount, sizeof(float4));
	hipHostMalloc((void**)&gsop.h_energies, size);
	int i;
	for(i = 0; i < gsop.aminoCount; i++){
		gsop.h_energies[i].x = 0.0f;
		gsop.h_energies[i].y = 0.0f;
		gsop.h_energies[i].z = 0.0f;
		gsop.h_energies[i].w = 0.0f;
	}
	hipMalloc((void**)&gsop.d_energies, size);
	//hipMalloc((void**)&gsop.d_energiesToSave, size);
	// Copying to the Device
	hipMemcpy(gsop.d_energies, gsop.h_energies, size, hipMemcpyHostToDevice);
	//hipMemcpy(gsop.d_energiesToSave, gsop.h_energies, size, hipMemcpyHostToDevice);
}

void bindTextures(){
#ifndef NOTEXTURE
	hipBindTexture(0, t_coord, gsop.d_coord, gsop.aminoCount*sizeof(float4));
#endif
}

void checkCUDAError(){
	hipError_t error = hipGetLastError();
	if(error != hipSuccess){
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(0);
	}
}
